#include "hip/hip_runtime.h"
/******************************************************************************
* FILE: 
* DESCRIPTION:
*
* GROUP INFO:
 pranjan            Pritesh Ranjan
 kmishra            Kushagra Mishra
 aapatel8           Akshit Patel
* 
* 
******************************************************************************/


#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include "mpilake.h"

#define __DEBUG

#define CUDA_CALL( err )     __cudaSafeCall( err, __FILE__, __LINE__ )
#define CUDA_CHK_ERR() __cudaCheckError(__FILE__,__LINE__)

/**************************************
* void __cudaSafeCall(hipError_t err, const char *file, const int line)
* void __cudaCheckError(const char *file, const int line)
*
* These routines were taken from the GPU Computing SDK
* (http://developer.nvidia.com/gpu-computing-sdk) include file "cutil.h"
**************************************/
inline void __cudaSafeCall( hipError_t err, const char *file, const int line ) {
#ifdef __DEBUG

#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
  do
  {
    if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
              file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
  } while ( 0 );
#pragma warning( pop )
#endif  // __DEBUG
  return;
}

inline void __cudaCheckError( const char *file, const int line ) {
#ifdef __DEBUG
#pragma warning( push )
#pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
  do
  {
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() failed at %s:%i : %s.\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }
    // More careful checking. However, this will affect performance.
    // Comment if not needed.
    /*err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
      fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s.\n",
               file, line, hipGetErrorString( err ) );
      exit( -1 );
    }*/
  } while ( 0 );
#pragma warning( pop )
#endif // __DEBUG
  return;
}

int tpdt_2(double *t, double dt, double tf) {
    if((*t) + dt > tf) return 0;
    (*t) = (*t) + dt;
    return 1;
}

__device__ double f_2(double p, double t) {
    return -expf(-TSCALE * t) * p;
}

void run_gpu(double *u, double *u0, double *u1, double *pebbles, int n, double h, double end_time, int nthreads)
{
	hipEvent_t kstart, kstop;
	float ktime;
    printf("\nGPU method called\n");
	/* HW2: Define your local variables here */
    
    /* Divide the (npoints * npoints) sized lake into 4 rectangular sized 
       parallel smaller lakes.
       Dividing this way maintains the continous addressing scheme of a 1D 
       array at a grid.
    */
    int npoints_y = npoints/GPU_COUNT;
    int npoints_x = npoints;
     
    int nblocks_y = npoints_y/nthreads;
    int nblocks_x = npoints_x/nthreads;
    
        /* Set up device timers */  
	CUDA_CALL(hipSetDevice(0));
	CUDA_CALL(hipEventCreate(&kstart));
	CUDA_CALL(hipEventCreate(&kstop));

	/* HW2: Add CUDA kernel call preperation code here */

	/* Start GPU computation timer */
	CUDA_CALL(hipEventRecord(kstart, 0));

	/* HW2: Add main lake simulation loop here */
	
        /* Stop GPU computation timer */
	CUDA_CALL(hipEventRecord(kstop, 0));
	CUDA_CALL(hipEventSynchronize(kstop));
	CUDA_CALL(hipEventElapsedTime(&ktime, kstart, kstop));
	printf("GPU computation: %f msec\n", ktime);

	/* HW2: Add post CUDA kernel call processing and cleanup here */

	/* timer cleanup */
    printf("\nGPU method called\n");

	CUDA_CALL(hipEventDestroy(kstart));
	CUDA_CALL(hipEventDestroy(kstop));
}
